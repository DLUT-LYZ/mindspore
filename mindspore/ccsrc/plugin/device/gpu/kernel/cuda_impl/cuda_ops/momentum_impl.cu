#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <iostream>
#include "momentum_impl.cuh"
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/elementswise_op_impl.cuh"

template <typename T, typename S, typename G>
struct MomentumUpdateVariableFunctor {
  const S *learning_rate_;
  const S *momentum_;
  MomentumUpdateVariableFunctor(const S *learning_rate, const S *momentum) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
  }
  __device__ __forceinline__ void operator()(T *variable, T *accumulation, const G *gradient) const {
    accumulation[0] = momentum_[0] * accumulation[0] + gradient[0];
    variable[0] -= learning_rate_[0] * accumulation[0];
  }
};

template <typename T, typename S, typename G>
struct MomentumUpdateVariableWithNesterovFunctor {
  const S *learning_rate_;
  const S *momentum_;
  MomentumUpdateVariableWithNesterovFunctor(const S *learning_rate, const S *momentum) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
  }
  __device__ __forceinline__ void operator()(T *variable, T *accumulation, const G *gradient) const {
    accumulation[0] = momentum_[0] * accumulation[0] + gradient[0];
    variable[0] -= gradient[0] * learning_rate_[0] + accumulation[0] * momentum_[0] * learning_rate_[0];
  }
};

template <>
struct MomentumUpdateVariableFunctor<half, float, half> {
  const float *learning_rate_;
  const float *momentum_;
  MomentumUpdateVariableFunctor(const float *learning_rate, const float *momentum) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
  }
  __device__ __forceinline__ void operator()(half *variable, half *accumulation, const half *gradient) const {
    accumulation[0] = __float2half(momentum_[0]) * accumulation[0] + gradient[0];
    variable[0] -= __float2half(learning_rate_[0]) * accumulation[0];
  }
};

template <>
struct MomentumUpdateVariableWithNesterovFunctor<half, float, half> {
  const float *learning_rate_;
  const float *momentum_;
  MomentumUpdateVariableWithNesterovFunctor(const float *learning_rate, const float *momentum) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
  }
  __device__ __forceinline__ void operator()(half *variable, half *accumulation, const half *gradient) const {
    accumulation[0] = __float2half(momentum_[0]) * accumulation[0] + gradient[0];
    variable[0] -= gradient[0] * __float2half(learning_rate_[0]) +
                   accumulation[0] * __float2half(momentum_[0]) * __float2half(learning_rate_[0]);
  }
};

template <>
struct MomentumUpdateVariableFunctor<float, float, half> {
  const float *learning_rate_;
  const float *momentum_;
  MomentumUpdateVariableFunctor(const float *learning_rate, const float *momentum) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
  }
  __device__ __forceinline__ void operator()(float *variable, float *accumulation, const half *gradient) const {
    accumulation[0] = momentum_[0] * accumulation[0] + __half2float(gradient[0]);
    variable[0] -= learning_rate_[0] * accumulation[0];
  }
};

template <>
struct MomentumUpdateVariableWithNesterovFunctor<float, float, half> {
  const float *learning_rate_;
  const float *momentum_;
  MomentumUpdateVariableWithNesterovFunctor(const float *learning_rate, const float *momentum) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
  }
  __device__ __forceinline__ void operator()(float *variable, float *accumulation, const half *gradient) const {
    accumulation[0] = momentum_[0] * accumulation[0] + __half2float(gradient[0]);
    variable[0] -= __half2float(gradient[0]) * learning_rate_[0] + accumulation[0] * momentum_[0] * learning_rate_[0];
  }
};

template <typename T, typename G>
struct FusedMomentumWeightDecayScaleFunctor {
  const T *learning_rate_;
  const T *momentum_;
  const T *weight_decay_;
  const T *scale_;
  FusedMomentumWeightDecayScaleFunctor(const T *learning_rate, const T *momentum, const T *weight_decay,
                                       const T *scale) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
    this->weight_decay_ = weight_decay;
    this->scale_ = scale;
  }
  __device__ __forceinline__ void operator()(T *variable, T *accumulation, const G *gradient) const {
    T grad = (variable[0] * weight_decay_[0] + static_cast<T>(gradient[0])) * scale_[0];
    accumulation[0] = momentum_[0] * accumulation[0] + grad;
    variable[0] -= learning_rate_[0] * accumulation[0];
  }
};

template <typename T, typename G>
struct FusedMomentumScaleFunctor {
  const T *learning_rate_;
  const T *momentum_;
  const T *scale_;
  FusedMomentumScaleFunctor(const T *learning_rate, const T *momentum, const T *scale) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
    this->scale_ = scale;
  }
  __device__ __forceinline__ void operator()(T *variable, T *accumulation, const G *gradient) const {
    accumulation[0] = momentum_[0] * accumulation[0] + static_cast<T>(gradient[0]) * scale_[0];
    variable[0] -= learning_rate_[0] * accumulation[0];
  }
};

template <typename T, typename G>
struct FusedWeightDecayMomentumFunctor {
  const T *learning_rate_;
  const T *momentum_;
  const T *weight_decay_;
  FusedWeightDecayMomentumFunctor(const T *learning_rate, const T *momentum, const T *weight_decay) {
    this->learning_rate_ = learning_rate;
    this->momentum_ = momentum;
    this->weight_decay_ = weight_decay;
  }
  __device__ __forceinline__ void operator()(T *variable, T *accumulation, const G *gradient) const {
    T grad = variable[0] * weight_decay_[0] + static_cast<T>(gradient[0]);
    accumulation[0] = momentum_[0] * accumulation[0] + grad;
    variable[0] -= learning_rate_[0] * accumulation[0];
  }
};

template <typename T, typename S, typename G>
void MomentumUpdateVariable(const size_t size, T *variable, T *accumulation, const S *learning_rate, const G *gradient,
                            const S *momentum, bool use_nesterov, hipStream_t hip_stream) {
  if (use_nesterov) {
    MomentumUpdateVariableWithNesterovFunctor<T, S, G> functor{learning_rate, momentum};
    cuda::elementwise::UnaryInputBinaryOutput(functor, (uint)(size), variable, accumulation, gradient, hip_stream);
  } else {
    MomentumUpdateVariableFunctor<T, S, G> functor{learning_rate, momentum};
    cuda::elementwise::UnaryInputBinaryOutput(functor, (uint)(size), variable, accumulation, gradient, hip_stream);
  }
}

template <typename T, typename G>
void FusedWeightDecayScaleMomentum(const size_t size, T *weight_decay, T *scale, T *variable, T *accumulation,
                                   const T *learning_rate, const G *gradient, const T *momentum,
                                   hipStream_t hip_stream) {
  FusedMomentumWeightDecayScaleFunctor<T, G> functor{learning_rate, momentum, weight_decay, scale};
  cuda::elementwise::UnaryInputBinaryOutput(functor, (uint)(size), variable, accumulation, gradient, hip_stream);
}

template <typename T, typename G>
void FusedScaleMomentum(const size_t size, T *scale, T *variable, T *accumulation, const T *learning_rate,
                        const G *gradient, const T *momentum, hipStream_t hip_stream) {
  FusedMomentumScaleFunctor<T, G> functor{learning_rate, momentum, scale};
  cuda::elementwise::UnaryInputBinaryOutput(functor, (uint)(size), variable, accumulation, gradient, hip_stream);
}

template <typename T, typename G>
void FusedWeightDecayMomentum(const size_t size, T *weight_decay, T *variable, T *accumulation, const T *learning_rate,
                              const G *gradient, const T *momentum, hipStream_t hip_stream) {
  FusedWeightDecayMomentumFunctor<T, G> functor{learning_rate, momentum, weight_decay};
  cuda::elementwise::UnaryInputBinaryOutput(functor, (uint)(size), variable, accumulation, gradient, hip_stream);
}

// CombineFusedScaleMomentum
template <typename T, typename S>
__global__ void CombineFusedMomentumScaleKernel(const size_t num, const size_t *element_num, T **scale, T **variable,
                                                T **accumulation, T **learning_rate, S **gradient, T **momentum) {
  for (size_t idx = 0; idx < num; idx++) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (element_num[idx]); i += blockDim.x * gridDim.x) {
      accumulation[idx][i] = momentum[idx][0] * accumulation[idx][i] + static_cast<T>(gradient[idx][i]) * scale[idx][0];
      variable[idx][i] -= learning_rate[idx][0] * accumulation[idx][i];
    }
  }
}

template <typename T, typename S>
void CombineFusedScaleMomentum(const size_t max, const size_t num, const size_t *elements, T **scale, T **variable,
                               T **accumulation, T **learning_rate, S **gradient, T **momentum,
                               hipStream_t hip_stream) {
  size_t thread_per_block = 256;
  size_t block_per_grid = (max + thread_per_block - 1) / thread_per_block;
  CombineFusedMomentumScaleKernel<<<block_per_grid, thread_per_block, 0, hip_stream>>>(
    num, elements, scale, variable, accumulation, learning_rate, gradient, momentum);
}
// end CombineFusedScaleMomentum

// CombineFusedWeightDecayScaleMomentum
template <typename T, typename S>
__global__ void CombineFusedMomentumWeightDecayScaleKernel(const size_t num, const size_t *element_num,
                                                           T **weight_decay, T **scale, T **variable, T **accumulation,
                                                           T **learning_rate, S **gradient, T **momentum) {
  for (size_t idx = 0; idx < num; idx++) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (element_num[idx]); i += blockDim.x * gridDim.x) {
      T grad = (variable[idx][i] * weight_decay[idx][0] + static_cast<T>(gradient[idx][i])) * scale[idx][0];
      accumulation[idx][i] = momentum[idx][0] * accumulation[idx][i] + grad;
      variable[idx][i] -= learning_rate[idx][0] * accumulation[idx][i];
    }
  }
}

template <typename T, typename S>
void CombineFusedWeightDecayScaleMomentum(const size_t max, const size_t num, const size_t *element_num,
                                          T **weight_decay, T **scale, T **variable, T **accumulation,
                                          T **learning_rate, S **gradient, T **momentum, hipStream_t hip_stream) {
  size_t thread_per_block = 256;
  size_t block_per_grid = (max + thread_per_block - 1) / thread_per_block;
  CombineFusedMomentumWeightDecayScaleKernel<<<block_per_grid, thread_per_block, 0, hip_stream>>>(
    num, element_num, weight_decay, scale, variable, accumulation, learning_rate, gradient, momentum);
}
// end CombineFusedWeightDecayScaleMomentum
template CUDA_LIB_EXPORT void MomentumUpdateVariable<float, float, float>(const size_t size, float *variable,
                                                                          float *accumulation,
                                                                          const float *learning_rate,
                                                                          const float *gradient, const float *momentum,
                                                                          bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<half, half, half>(const size_t size, half *variable,
                                                                       half *accumulation, const half *learning_rate,
                                                                       const half *gradient, const half *momentum,
                                                                       bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<half, float, half>(const size_t size, half *variable,
                                                                        half *accumulation, const float *learning_rate,
                                                                        const half *gradient, const float *momentum,
                                                                        bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<float, float, half>(const size_t size, float *variable,
                                                                         float *accumulation,
                                                                         const float *learning_rate,
                                                                         const half *gradient, const float *momentum,
                                                                         bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<int8_t, int8_t, int8_t>(
  const size_t size, int8_t *variable, int8_t *accumulation, const int8_t *learning_rate, const int8_t *gradient,
  const int8_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<uint8_t, uint8_t, uint8_t>(
  const size_t size, uint8_t *variable, uint8_t *accumulation, const uint8_t *learning_rate, const uint8_t *gradient,
  const uint8_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<int16_t, int16_t, int16_t>(
  const size_t size, int16_t *variable, int16_t *accumulation, const int16_t *learning_rate, const int16_t *gradient,
  const int16_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<uint16_t, uint16_t, uint16_t>(
  const size_t size, uint16_t *variable, uint16_t *accumulation, const uint16_t *learning_rate,
  const uint16_t *gradient, const uint16_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<uint32_t, uint32_t, uint32_t>(
  const size_t size, uint32_t *variable, uint32_t *accumulation, const uint32_t *learning_rate,
  const uint32_t *gradient, const uint32_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<int32_t, int32_t, int32_t>(
  const size_t size, int32_t *variable, int32_t *accumulation, const int32_t *learning_rate, const int32_t *gradient,
  const int32_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<int64_t, int64_t, int64_t>(
  const size_t size, int64_t *variable, int64_t *accumulation, const int64_t *learning_rate, const int64_t *gradient,
  const int64_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<uint64_t, uint64_t, uint64_t>(
  const size_t size, uint64_t *variable, uint64_t *accumulation, const uint64_t *learning_rate,
  const uint64_t *gradient, const uint64_t *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<double, double, double>(
  const size_t size, double *variable, double *accumulation, const double *learning_rate, const double *gradient,
  const double *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<Complex<float>, Complex<float>, Complex<float>>(
  const size_t size, Complex<float> *variable, Complex<float> *accumulation, const Complex<float> *learning_rate,
  const Complex<float> *gradient, const Complex<float> *momentum, bool use_nesterov, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void MomentumUpdateVariable<Complex<double>, Complex<double>, Complex<double>>(
  const size_t size, Complex<double> *variable, Complex<double> *accumulation, const Complex<double> *learning_rate,
  const Complex<double> *gradient, const Complex<double> *momentum, bool use_nesterov, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void FusedWeightDecayScaleMomentum(const size_t element_num, float *weight_decay, float *scale,
                                                            float *variable, float *accumulation,
                                                            const float *learning_rate, const float *gradient,
                                                            const float *momentum, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FusedWeightDecayScaleMomentum(const size_t element_num, float *weight_decay, float *scale,
                                                            float *variable, float *accumulation,
                                                            const float *learning_rate, const half *gradient,
                                                            const float *momentum, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FusedWeightDecayMomentum(const size_t element_num, float *weight_decay, float *variable,
                                                       float *accumulation, const float *learning_rate,
                                                       const float *gradient, const float *momentum,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FusedWeightDecayMomentum(const size_t element_num, float *weight_decay, float *variable,
                                                       float *accumulation, const float *learning_rate,
                                                       const half *gradient, const float *momentum,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FusedScaleMomentum(const size_t element_num, float *scale, float *variable,
                                                 float *accumulation, const float *learning_rate, const float *gradient,
                                                 const float *momentum, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FusedScaleMomentum(const size_t element_num, float *scale, float *variable,
                                                 float *accumulation, const float *learning_rate, const half *gradient,
                                                 const float *momentum, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CombineFusedWeightDecayScaleMomentum(
  const size_t max, const size_t num, const size_t *elements, float **weight_decay, float **scale, float **variable,
  float **accumulation, float **learning_rate, float **gradient, float **momentum, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CombineFusedWeightDecayScaleMomentum(
  const size_t max, const size_t num, const size_t *elements, float **weight_decay, float **scale, float **variable,
  float **accumulation, float **learning_rate, half **gradient, float **momentum, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CombineFusedScaleMomentum(const size_t max, const size_t num, const size_t *elements,
                                                        float **scale, float **variable, float **accumulation,
                                                        float **learning_rate, float **gradient, float **momentum,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CombineFusedScaleMomentum(const size_t max, const size_t num, const size_t *elements,
                                                        float **scale, float **variable, float **accumulation,
                                                        float **learning_rate, half **gradient, float **momentum,
                                                        hipStream_t hip_stream);
