#include "hip/hip_runtime.h"
/**
 * Copyright 2023 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include <limits>
#include <algorithm>

#include "include/hip/hip_runtime.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/reduce_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/permutation_in_iterator.cuh"

constexpr int thread_per_warp = 32;
constexpr int kUnroll = 8;

template <typename T>
struct Sum {
  __host__ __device__ T operator()(const T &a, const T &b) const { return a + b; }
};

template <typename T>
struct Max {
  __host__ __device__ T operator()(const T &a, const T &b) const { return (a != a ? a : (a > b ? a : b)); }
};

template <typename T>
struct Min {
  __host__ __device__ T operator()(const T &a, const T &b) const { return (a != a ? a : (a < b ? a : b)); }
};

template <typename T>
struct Prod {
  __host__ __device__ T operator()(const T &a, const T &b) const { return a * b; }
};

struct And {
  __host__ __device__ bool operator()(const bool &a, const bool &b) const { return a && b; }
};

struct Or {
  __host__ __device__ bool operator()(const bool &a, const bool &b) const { return a || b; }
};

template <typename T, typename Op>
struct IsSum {
  constexpr static bool flag = std::is_same<Op, Sum<T>>::value;
};

template <typename T, typename Op>
struct IsProd {
  constexpr static bool flag = std::is_same<Op, Prod<T>>::value;
};

template <typename T, typename Op>
struct IsMax {
  constexpr static bool flag = std::is_same<Op, Max<T>>::value;
};

template <typename T, typename Op>
struct IsMin {
  constexpr static bool flag = std::is_same<Op, Min<T>>::value;
};

template <typename Op>
struct IsAll {
  constexpr static bool flag = std::is_same<Op, And>::value;
};

template <typename Op>
struct IsAny {
  constexpr static bool flag = std::is_same<Op, Or>::value;
};

template <typename T, typename Op>
struct GetInit {
  static_assert(IsSum<T, Op>::flag || IsProd<T, Op>::flag || IsMax<T, Op>::flag || IsMin<T, Op>::flag ||
                  IsAll<Op>::flag || IsAny<Op>::flag,
                "Not support this type");

  template <typename U = T, typename OpCopy = Op>
  U operator()(typename std::enable_if<IsSum<U, OpCopy>::flag, U>::type init = U(0)) {
    return init;
  }

  template <typename U = T, typename OpCopy = Op>
  U operator()(typename std::enable_if<IsProd<U, OpCopy>::flag, U>::type init = U(1)) {
    return init;
  }

  template <typename U = T, typename OpCopy = Op>
  U operator()(typename std::enable_if<IsMax<U, OpCopy>::flag, U>::type init = std::numeric_limits<T>::lowest()) {
    return init;
  }

  template <typename U = T, typename OpCopy = Op>
  U operator()(typename std::enable_if<IsMin<U, OpCopy>::flag, U>::type init = std::numeric_limits<T>::max()) {
    return init;
  }

  template <typename U = T, typename OpCopy = Op>
  U operator()(typename std::enable_if<IsAll<OpCopy>::flag, bool>::type init = true) {
    return init;
  }

  template <typename U = T, typename OpCopy = Op>
  U operator()(typename std::enable_if<IsAny<OpCopy>::flag, bool>::type init = false) {
    return init;
  }
};

template <typename T>
__global__ void Average(const size_t size, const size_t divisor, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output[pos] /= divisor;
  }
}

template <>
__global__ void Average(const size_t size, const size_t divisor, half *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
#if CUDA_VERSION >= 11000
    output[pos] /= __ull2half_rn(divisor);
#else
    output[pos] /= static_cast<half>(static_cast<float>(divisor));
#endif  // CUDA_VERSION > 11000
  }
}

template <>
__global__ void Average(const size_t size, const size_t divisor, Complex<float> *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    Complex<float> ret;
    ret.real(output[pos].real() / divisor);
    ret.imag(output[pos].imag() / divisor);
    output[pos] = ret;
  }
}

template <>
__global__ void Average(const size_t size, const size_t divisor, Complex<double> *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    Complex<double> ret;
    ret.real(output[pos].real() / divisor);
    ret.imag(output[pos].imag() / divisor);
    output[pos] = ret;
  }
}

struct GatherOp {
  __host__ __device__ GatherOp(const int &extent_x, const int &extent_y, const int &extent_z, bool kOne)
      : extent_x_(extent_x), extent_y_(extent_y), extent_z_(extent_z), kOne_(kOne) {
    if (kOne_)
      group_size_ = extent_y_;
    else
      group_size_ = extent_x_ * extent_z_;
  }

  __host__ __device__ int operator()(const int &ind) const {
    const int group = kOne_ ? ind / group_size_ : ind % group_size_;
    const int offset = kOne_ ? ind % group_size_ : ind / group_size_;

    const int x = group / extent_z_;
    const int z = group % extent_z_;

    return x * extent_y_ * extent_z_ + z + offset * extent_z_;
  }

  int extent_x_;
  int extent_y_;
  int extent_z_;
  bool kOne_;
  int group_size_;
};

struct ComputeOffset {
  __host__ __device__ explicit ComputeOffset(const int &cols) : cols_(cols) {}

  __host__ __device__ int operator()(const int &x) const { return cols_ * x; }

  int cols_;
};

template <typename T, int NUM_THREADS, typename Op>
__global__ __launch_bounds__(1024) void BlockReduceKernel(const T *input, T *output, const size_t size, Op op,
                                                          const T init) {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  const int gid = blockDim.x * bid + tid;
  const int stride = blockDim.x * gridDim.x;
  T sum = init;
  if (gid < size) {
    sum = input[gid];
    for (size_t i = gid + stride; i < size; i += stride) {
      sum = op(sum, input[i]);
    }
  }
  typedef hipcub::BlockReduce<T, NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const int num_elem_need_reduce =
    max(min(static_cast<float>(size - bid * blockDim.x), static_cast<float>(NUM_THREADS)), static_cast<float>(0.0));
  sum = BlockReduce(temp_storage).Reduce(sum, op, num_elem_need_reduce);
  if (tid == 0) output[bid] = sum;
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void CleanupSegments(const T *temp, T *output, const size_t num_rows,
                                                        const size_t num_cols, const size_t size, Op op, T init) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  T temp_res = init;
  if (tid < size * num_cols) {
    temp_res = temp[tid];
  }
  typedef hipcub::WarpReduce<T> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage;
  const bool head_flag = (threadIdx.x % size) == 0;
  T sum = WarpReduce(temp_storage).HeadSegmentedReduce(temp_res, head_flag, op);
  if (head_flag && tid < size * num_cols) {
    output[tid / size] = sum;
  }
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void RowReduceKernel(const T *input, T *output, const size_t num_rows,
                                                        const size_t num_cols, Op op, T init) {
  CUDA_KERNEL_ASSERT(blockDim.x % thread_per_warp == 0);
  int warp_per_block = blockDim.x / thread_per_warp;
  int warp_index = threadIdx.x / thread_per_warp;
  const int row_index = blockIdx.x * warp_per_block + warp_index;
  const int lane_index = threadIdx.x % thread_per_warp;

  if (num_cols == 1) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < num_rows) {
      output[gid] = input[gid];
    }
    return;
  }
  T sum = init;
  int col_index = lane_index;
  if (row_index < num_rows && col_index < num_cols) {
    sum = input[row_index * num_cols + col_index];
    col_index += thread_per_warp;
    for (; col_index < num_cols; col_index += thread_per_warp) {
      sum = op(sum, input[row_index * num_cols + col_index]);
    }
  }

  typedef hipcub::WarpReduce<T> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage;
  sum = WarpReduce(temp_storage).Reduce(sum, op, min(static_cast<int>(num_cols), thread_per_warp));
  if (row_index < num_rows && lane_index == 0) {
    output[row_index] = sum;
  }
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void ColumnReduce16Kernel(const T *input, T *output, const size_t num_rows,
                                                             const size_t num_cols, Op op, T init) {
  const int rows_per_warp = thread_per_warp / num_cols;

  const int lane_index = threadIdx.x % thread_per_warp;
  const int lane_row = lane_index / num_cols;

  const int start_row_warp = rows_per_warp * (blockIdx.y * blockDim.y + threadIdx.y);
  const int start_row_lane = start_row_warp + lane_row;
  int row_index = start_row_lane;
  int col_index = lane_index % num_cols;

  T sum = init;
  if (row_index * num_cols + col_index < num_rows * num_cols) sum = input[row_index * num_cols + col_index];
  __shared__ __align__(alignof(T)) char partial_sums_raw[thread_per_warp * (thread_per_warp + 1) * sizeof(T)];
  T *partial_sums = reinterpret_cast<T *>(partial_sums_raw);

  row_index += rows_per_warp * gridDim.y * blockDim.y;
  for (; row_index < num_rows; row_index += rows_per_warp * gridDim.y * blockDim.y) {
    int global_pos = row_index * num_cols + col_index;
    if (global_pos < (num_rows * num_cols)) sum = op(sum, input[row_index * num_cols + col_index]);
  }

  const int rows_in_this_warp = min(rows_per_warp, static_cast<int>(num_rows - start_row_warp));
  for (int i = 1; i < rows_in_this_warp; ++i) {
    T tmp = hipcub::ShuffleIndex<thread_per_warp, T>(sum, static_cast<int>(threadIdx.x + i * num_cols), 0xffffffff);
    if (lane_index < num_cols) sum = op(sum, tmp);
  }

  if (lane_index < num_cols) partial_sums[lane_index * (thread_per_warp + 1) + threadIdx.y] = sum;

  __syncthreads();

  if (threadIdx.y == 0 && threadIdx.x < num_cols) {
    T total_sum = partial_sums[threadIdx.x * (thread_per_warp + 1)];

    if (blockDim.y > 1) {
      for (int row_index = 1; row_index < blockDim.y; ++row_index) {
        T block_sum = partial_sums[threadIdx.x * (thread_per_warp + 1) + row_index];
        total_sum = op(total_sum, block_sum);
      }
    }

    output[col_index * gridDim.y + blockIdx.y] = total_sum;
  }
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void ColumnReduceKernel(const T *input, T *output, const size_t num_rows,
                                                           const size_t num_cols, Op op, T init) {
  size_t row_index = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col_index = blockIdx.x * thread_per_warp + threadIdx.x;

  T sum = init;
  if (row_index < num_rows && col_index < num_cols) {
    sum = input[row_index * num_cols + col_index];
  }

  __shared__ __align__(alignof(T)) char partial_sums_raw[thread_per_warp * (thread_per_warp + 1) * sizeof(T)];
  T *partial_sums = reinterpret_cast<T *>(partial_sums_raw);

  row_index += gridDim.y * blockDim.y;

  if (col_index < num_cols) {
    for (; row_index < num_rows; row_index += gridDim.y * blockDim.y) {
      sum = op(sum, input[row_index * num_cols + col_index]);
    }
  }

  partial_sums[threadIdx.x * (thread_per_warp + 1) + threadIdx.y] = sum;

  __syncthreads();

  if (threadIdx.y == 0 && col_index < num_cols) {
    T total_sum = partial_sums[threadIdx.x * (thread_per_warp + 1)];
    const int numRowsThisBlock =
      min(static_cast<int>(blockDim.y), static_cast<int>(num_rows - blockIdx.y * blockDim.y));
    for (int row_index = 1; row_index < numRowsThisBlock; ++row_index) {
      T block_sum = partial_sums[threadIdx.x * (thread_per_warp + 1) + row_index];
      total_sum = op(total_sum, block_sum);
    }
    output[col_index * gridDim.y + blockIdx.y] = total_sum;
  }
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void ColumnReduceSimpleKernel(const T *input, T *output, const size_t num_matrix,
                                                                 const size_t num_rows, const size_t num_cols, Op op) {
  const int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const int matrix_size = num_rows * num_cols;

  const int matrix_index = gid / num_cols;
  const int col_index = gid % num_cols;

  if (matrix_index >= num_matrix) return;

  if (num_rows == 1) {
    output[matrix_index * matrix_size + col_index] = input[matrix_index * matrix_size + col_index];
    return;
  }

  T sum = op(input[matrix_index * matrix_size + col_index], input[matrix_index * matrix_size + num_cols + col_index]);
  for (int row_index = 2; row_index < num_rows; ++row_index) {
    sum = op(sum, input[matrix_index * matrix_size + row_index * num_cols + col_index]);
  }

  output[matrix_index * num_cols + col_index] = sum;
}

template <typename T, typename Op>
__device__ __inline__ T ComputeSum(const T *input, const int plane, const int num_out_rows, int num_rows, int num_cols,
                                   const int col, Op op) {
  const int out_rows = num_rows / (2 * kUnroll);
  const int num_rem_rows = num_rows % (2 * kUnroll);
  const int elems_per_plane = num_rows * num_cols;
  T reg[2 * kUnroll];
  T sum;
  int offset = 0;
  if (out_rows != 0) {
    for (int i = 0; i < 2 * kUnroll; i++) {
      reg[i] = input[plane * elems_per_plane + i * (num_out_rows * num_cols) + col];
    }
    sum = reg[0];
    for (int i = 1; i < 2 * kUnroll; i++) {
      sum = op(sum, reg[i]);
    }
    offset = 2 * kUnroll * (num_out_rows * num_cols);
  }

  if (col < num_cols && num_rem_rows > 0) {
    reg[0] = input[plane * elems_per_plane + offset + 0 * num_cols + col];
    if (out_rows != 0) {
      sum = op(sum, reg[0]);
    } else {
      sum = reg[0];
    }
    for (int i = 1; i < num_rem_rows; i++) {
      reg[0] = input[plane * elems_per_plane + offset + i * num_cols + col];
      sum = op(sum, reg[0]);
    }
  }
  return sum;
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void ColumnReduceInToTempKernel(T *temp, int temp_in_offset, int temp_out_offset,
                                                                   const T *input, const size_t num_planes,
                                                                   int num_rows, const size_t num_cols, Op op) {
  T *t = reinterpret_cast<T *>(temp);
  T *out_ = t + temp_out_offset;

  const int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_out_rows = max(1, num_rows / (2 * kUnroll));
  const int plane = gid / (num_out_rows * num_cols);
  const int col = gid % (num_out_rows * num_cols);

  if (plane >= num_planes) return;

  T sum;
  if (temp_in_offset == -1) {
    auto in_ = input;
    sum = ComputeSum(in_, plane, num_out_rows, num_rows, num_cols, col, op);
  } else {
    auto in_ = t + temp_in_offset;
    sum = ComputeSum(in_, plane, num_out_rows, num_rows, num_cols, col, op);
  }
  out_[plane * num_out_rows * num_cols + col] = sum;
}

template <typename T, typename Op>
__global__ __launch_bounds__(1024) void ColumnReduceTempToOutKernel(T *temp, int temp_in_offset, const T *input,
                                                                    T *output, const size_t num_planes, int num_rows,
                                                                    const size_t num_cols, Op op) {
  T *t = temp;
  const int tid = threadIdx.x;
  const int gid = threadIdx.x + blockIdx.x * blockDim.x;
  int elems_per_plane = num_rows * num_cols;

  if (num_rows == 1) {
    if (gid >= num_planes * num_cols) return;
    if (temp_in_offset == -1) {
      auto in_ = input;
      output[gid] = in_[gid];
    } else {
      auto in_ = t + temp_in_offset;
      output[gid] = in_[gid];
    }
    return;
  }

  const int planes_per_block = 1;
  const int plane = blockIdx.x * planes_per_block + tid / elems_per_plane;
  const int col = tid % elems_per_plane;
  const int local_plane = plane % planes_per_block;

  if (tid >= planes_per_block * elems_per_plane || plane >= num_planes) return;

  extern __shared__ __align__(8) char ss[];
  T *smem = reinterpret_cast<T *>(ss);

  if (temp_in_offset == -1) {
    auto in_ = input;
    smem[local_plane * elems_per_plane + col] = in_[plane * elems_per_plane + col];
  } else {
    auto in_ = t + temp_in_offset;
    smem[local_plane * elems_per_plane + col] = in_[plane * elems_per_plane + col];
  }
  __syncthreads();

  int num_in_rows = num_rows;
  int num_out_rows;
  int num_rem_rows;

  int in_offset = 0;
  int out_offset = blockDim.x;

  int in_elems_per_plane = elems_per_plane;
  int out_elems_per_plane;

  while (num_in_rows > 1) {
    num_out_rows = num_in_rows / 2;
    num_rem_rows = num_in_rows % 2;
    out_elems_per_plane = num_out_rows * num_cols;

    if (col < out_elems_per_plane) {
      T sum;
      sum = op(smem[in_offset + local_plane * in_elems_per_plane + col],
               smem[in_offset + local_plane * in_elems_per_plane + out_elems_per_plane + col]);
      if (num_rem_rows == 1 && col < num_cols) {
        sum = op(sum, smem[in_offset + local_plane * in_elems_per_plane + 2 * out_elems_per_plane + col]);
      }
      smem[out_offset + local_plane * out_elems_per_plane + col] = sum;
    }

    num_in_rows = num_out_rows;
    in_elems_per_plane = out_elems_per_plane;
    int t_offset = in_offset;
    in_offset = out_offset;
    out_offset = t_offset;
    __syncthreads();
  }

  if (col < num_cols) {
    output[plane * num_cols + col] = smem[in_offset + local_plane * out_elems_per_plane + col];
  }
}

template <typename T, typename Op>
void CalReduceColumn16(const T *input, const size_t num_rows, const size_t num_cols, Op op, T init, T *output,
                       hipStream_t hip_stream) {
  int rows_per_warp = thread_per_warp / num_cols;
  const int block_y = std::min<int>(((num_rows + rows_per_warp - 1) / rows_per_warp), (1024 / thread_per_warp));

  dim3 num_threads(thread_per_warp, block_y, 1);

  const int grid_y = (num_rows + rows_per_warp * num_threads.y - 1) / (rows_per_warp * num_threads.y);
  dim3 num_blocks(1, grid_y, 1);

  num_blocks.y = std::min<int>(static_cast<int>(num_blocks.y), thread_per_warp);

  if (num_blocks.y > 2 && num_blocks.y < thread_per_warp) {
    int log2 = Log2Floor(num_blocks.y);
    num_blocks.y = 1 << log2;
  }

  if (num_blocks.y == 1) {
    ColumnReduce16Kernel<<<num_blocks, num_threads, 0, hip_stream>>>(input, output, num_rows, num_cols, op, init);
  } else {
    T *temp_storage = nullptr;
    (void)hipMalloc(&temp_storage, sizeof(T) * num_cols * num_blocks.y);
    ColumnReduce16Kernel<<<num_blocks, num_threads, 0, hip_stream>>>(input, temp_storage, num_rows, num_cols, op,
                                                                      init);

    const int grid_x = (num_blocks.y * num_cols + thread_per_warp - 1) / thread_per_warp;
    dim3 new_num_blocks(grid_x, 1, 1);
    dim3 new_num_threads(128, 1, 1);
    CleanupSegments<<<new_num_blocks, new_num_threads, 0, hip_stream>>>(temp_storage, output, num_rows, num_cols,
                                                                         num_blocks.y, op, init);
    (void)hipFree(temp_storage);
  }
}

template <typename T, typename Op>
void CalReduceColumn4096(const T *input, const size_t num_rows, const size_t num_cols, Op op, T init, T *output,
                         hipStream_t hip_stream) {
  dim3 num_threads(thread_per_warp, std::min<int>(num_rows, (1024 / thread_per_warp)), 1);
  dim3 num_blocks((num_cols + thread_per_warp - 1) / thread_per_warp, 1, 1);

  if (num_blocks.x < 16) {
    num_blocks.y = std::min<int>((num_rows + thread_per_warp - 1) / thread_per_warp, thread_per_warp);
  }
  if (num_blocks.y > 2 && num_blocks.y < thread_per_warp) {
    int log2 = Log2Floor(num_blocks.y);
    num_blocks.y = 1 << log2;
  }

  if (num_blocks.y == 1) {
    ColumnReduceKernel<<<num_blocks, num_threads, 0, hip_stream>>>(input, output, num_rows, num_cols, op, init);
  } else {
    T *temp_storage = nullptr;
    (void)hipMalloc(&temp_storage, sizeof(T) * num_cols * num_blocks.y);
    ColumnReduceKernel<<<num_blocks, num_threads, 0, hip_stream>>>(input, temp_storage, num_rows, num_cols, op, init);

    dim3 new_num_blocks((num_blocks.y * num_cols + thread_per_warp - 1) / thread_per_warp, 1, 1);
    CleanupSegments<<<new_num_blocks, num_threads, 0, hip_stream>>>(temp_storage, output, num_rows, num_cols,
                                                                     num_blocks.y, op, init);
    (void)hipFree(temp_storage);
  }
}

template <typename T, typename Op>
void CalReduceToScalar(const T *input, const size_t size, T *output, Op op, T init, hipStream_t hip_stream) {
  if (size <= 4096) {
    const int num_blocks = 1;
    const int num_threads = 256;
    BlockReduceKernel<T, num_threads, Op><<<num_blocks, num_threads, 0, hip_stream>>>(input, output, size, op, init);
    return;
  } else if (size <= 1 << 18) {
    const int num_threads = 256;
    const int num_blocks = std::min<int>(thread_per_warp, ((static_cast<int>(size) + num_threads - 1) / num_threads));
    T *temp_storage = nullptr;
    (void)hipMalloc(&temp_storage, num_blocks * sizeof(T));
    BlockReduceKernel<T, num_threads, Op>
      <<<num_blocks, num_threads, 0, hip_stream>>>(input, temp_storage, size, op, init);
    const int last_blocks = 1;
    const int num_rows = 1;
    const int num_cols = 1;
    CleanupSegments<<<last_blocks, thread_per_warp, 0, hip_stream>>>(temp_storage, output, num_rows, num_cols,
                                                                      num_blocks, op, init);
    hipFree(temp_storage);
    return;
  }
  size_t temp_storage_size = 0;
  auto reduce = [&](void *temp_storage) {
    auto res = hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_size, input, output, size, op, init, hip_stream);
    if (res != hipSuccess) {
      return;
    }
  };
  reduce(nullptr);
  T *temp_storage;
  (void)hipMalloc(&temp_storage, temp_storage_size);
  reduce(temp_storage);
  hipFree(temp_storage);
}

template <typename T, typename Op>
void CalReduceRow(const T *input, const size_t num_rows, const size_t num_cols, Op op, T init, T *output,
                  hipStream_t hip_stream) {
  if (num_cols < 1024) {
    const int num_threads = 128;
    const int num_warps = num_threads / thread_per_warp;
    const int num_blocks = (num_rows + num_warps - 1) / num_warps;
    RowReduceKernel<<<num_blocks, num_threads, 0, hip_stream>>>(input, output, num_rows, num_cols, op, init);
    return;
  }
  ComputeOffset computeoffset(num_cols);
  hipcub::CountingInputIterator<int> counting_iter(0);
  hipcub::TransformInputIterator<int, ComputeOffset, hipcub::CountingInputIterator<int>> transform_iter(counting_iter,
                                                                                                  computeoffset);
  size_t temp_storage_size = 0;
  auto reduce = [&](void *temp_storage) {
    auto res = hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, input, output, num_rows,
                                                  transform_iter, transform_iter + 1, op, init, hip_stream);
    if (res != hipSuccess) {
      return;
    }
  };

  reduce(nullptr);
  T *temp_storage = nullptr;
  (void)hipMalloc(&temp_storage, temp_storage_size);

  reduce(temp_storage);
  hipFree(temp_storage);
}

template <typename T, typename Op>
void CalReduceColumn(const T *input, const size_t dim0, const size_t dim1, Op op, T init, T *output,
                     hipStream_t hip_stream) {
  if (dim1 <= 16) {
    CalReduceColumn16(input, dim0, dim1, op, init, output, hip_stream);
  } else if (dim1 <= 4096) {
    CalReduceColumn4096(input, dim0, dim1, op, init, output, hip_stream);
  } else {
    const int num_threads = 128;
    const int num_blocks = (dim1 + num_threads - 1) / num_threads;
    const size_t num_matrix = 1;
    ColumnReduceSimpleKernel<<<num_blocks, num_threads, 0, hip_stream>>>(input, output, num_matrix, dim0, dim1, op);
  }
}

template <typename T, typename Op>
void CalReduce3DXZ(const T *input, const size_t dim0, const size_t dim1, const size_t dim2, Op op, T init, T *output,
                   hipStream_t hip_stream) {
  ComputeOffset computeoffset(dim0 * dim2);
  hipcub::CountingInputIterator<int> counting_iter(0);
  hipcub::TransformInputIterator<int, ComputeOffset, hipcub::CountingInputIterator<int>> transform_iter(counting_iter,
                                                                                                  computeoffset);
  GatherOp gather_op(dim0, dim1, dim2, false);
  typedef hipcub::TransformInputIterator<int, GatherOp, hipcub::CountingInputIterator<int>> gatherIterType;
  gatherIterType gather_iter(counting_iter, gather_op);
  PermutationInputIterator<T, gatherIterType> permute_iter(input, gather_iter);
  std::size_t temp_storage_size = 0;
  auto reduce = [&](void *temp_storage_ptr) {
    auto res = hipcub::DeviceSegmentedReduce::Reduce(temp_storage_ptr, temp_storage_size, permute_iter, output, dim1,
                                                  transform_iter, transform_iter + 1, op, init, hip_stream);
    if (res != hipSuccess) {
      return;
    }
  };

  reduce(nullptr);
  T *temp_storage = nullptr;
  (void)hipMalloc(&temp_storage, temp_storage_size);
  reduce(temp_storage);
  hipFree(temp_storage);
}

template <typename T, typename Op>
void CalReduce3DY(const T *input, const size_t dim0, const size_t dim1, const size_t dim2, Op op, T init, T *output,
                  hipStream_t hip_stream) {
  int num_threads = 128;
  int n_group_in = dim1;
  int n_size = dim2;

  std::size_t temp_storage_size = 0;
  while (n_group_in >= 2 && n_group_in * n_size > num_threads) {
    int n_group_out = std::max(1, n_group_in / (2 * kUnroll));
    temp_storage_size += n_group_out * n_size;
    n_group_in = n_group_out;
  }
  temp_storage_size *= dim0 * sizeof(T);
  T *temp_storage = nullptr;
  (void)hipMalloc(&temp_storage, temp_storage_size);

  n_group_in = dim1;
  int temp_in_offset = -1;
  int temp_out_offset = 0;
  int num_blocks;
  while (n_group_in >= 2 && n_group_in * n_size > num_threads) {
    int n_group_out = std::max(1, n_group_in / (2 * kUnroll));
    num_blocks = (static_cast<int>(dim0) * n_group_out * n_size + num_threads - 1) / num_threads;
    ColumnReduceInToTempKernel<<<num_blocks, num_threads, 0, hip_stream>>>(
      temp_storage, temp_in_offset, temp_out_offset, input, dim0, n_group_in, dim2, op);
    n_group_in = n_group_out;
    temp_in_offset = temp_out_offset;
    temp_out_offset = temp_in_offset + dim0 * n_group_out * n_size;
  }

  if (n_group_in * n_size <= num_threads) {
    num_blocks = dim0;
  } else if (n_group_in != 1) {
    return;
  } else {
    num_blocks = (static_cast<int>(dim0) * n_size + num_threads - 1) / num_threads;
  }
  ColumnReduceTempToOutKernel<<<num_blocks, num_threads, 2 * sizeof(T) * num_threads, hip_stream>>>(
    temp_storage, temp_in_offset, input, output, dim0, n_group_in, dim2, op);
  hipFree(temp_storage);
}

template <typename T, typename Op>
void CalReduce3DYLight(const T *input, const size_t dim0, const size_t dim1, const size_t dim2, Op op, T init,
                       T *output, hipStream_t hip_stream) {
  int threads_per_block = 128;
  int num_blocks = (dim0 * dim2 + threads_per_block - 1) / threads_per_block;
  ColumnReduceSimpleKernel<<<num_blocks, threads_per_block, 0, hip_stream>>>(input, output, dim0, dim1, dim2, op);
}

template <typename T, typename Op>
void ReduceImpl(const T *input, const std::vector<size_t> &input_reshape, const bool reduce_first_axis, Op op,
                T *output, hipStream_t hip_stream) {
  T init = GetInit<T, Op>()();
  const size_t dim0 = input_reshape[0];
  const size_t dim1 = input_reshape.size() >= 2 ? input_reshape[1] : 1;
  const size_t dim2 = input_reshape.size() >= 3 ? input_reshape[2] : 1;
  if (input_reshape.size() == 1 && reduce_first_axis) {
    CalReduceToScalar(input, dim0, output, op, init, hip_stream);
  } else if ((input_reshape.size() == 2) && (reduce_first_axis)) {
    CalReduceColumn(input, dim0, dim1, op, init, output, hip_stream);
  } else if ((input_reshape.size() == 2) && (!reduce_first_axis)) {
    CalReduceRow(input, dim0, dim1, op, init, output, hip_stream);
  } else if ((input_reshape.size() == 3) && (reduce_first_axis)) {
    CalReduce3DXZ(input, dim0, dim1, dim2, op, init, output, hip_stream);
  } else if ((input_reshape.size() == 3) && (!reduce_first_axis)) {
    int num_per_thread = dim1 / (dim0 * dim2);
    if (num_per_thread >= 16) {
      CalReduce3DY(input, dim0, dim1, dim2, op, init, output, hip_stream);
    } else {
      CalReduce3DYLight(input, dim0, dim1, dim2, op, init, output, hip_stream);
    }
  } else {
    fprintf(stderr, "Invalid shapes and axis to reduce.");
    exit(1);
  }
}

template <typename T>
hipError_t ArrayReduce(const T *input, const std::vector<size_t> &input_reshape, const bool reduce_first_axis,
                        ReduceType_t type, T *output, hipStream_t hip_stream) {
  switch (type) {
    case ReduceSum:
      ReduceImpl<T, Sum<T>>(input, input_reshape, reduce_first_axis, Sum<T>(), output, hip_stream);
      break;
    case ReduceMax:
      ReduceImpl<T, Max<T>>(input, input_reshape, reduce_first_axis, Max<T>(), output, hip_stream);
      break;
    case ReduceMin:
      ReduceImpl<T, Min<T>>(input, input_reshape, reduce_first_axis, Min<T>(), output, hip_stream);
      break;
    case ReduceProd:
      ReduceImpl<T, Prod<T>>(input, input_reshape, reduce_first_axis, Prod<T>(), output, hip_stream);
      break;
    case ReduceAll:
      ReduceImpl<T, And>(input, input_reshape, reduce_first_axis, And(), output, hip_stream);
      break;
    case ReduceAny:
      ReduceImpl<T, Or>(input, input_reshape, reduce_first_axis, Or(), output, hip_stream);
      break;
    case ReduceMean:
      size_t reduce_size = 1;
      size_t unreduce_size = 1;
      if (input_reshape.size() == 1) {
        reduce_size = input_reshape[0];
        unreduce_size = 1;
      } else if ((input_reshape.size() == 2) && (reduce_first_axis)) {
        reduce_size = input_reshape[0];
        unreduce_size = input_reshape[1];
      } else if ((input_reshape.size() == 2) && (!reduce_first_axis)) {
        reduce_size = input_reshape[1];
        unreduce_size = input_reshape[0];
      } else if ((input_reshape.size() == 3) && (reduce_first_axis)) {
        reduce_size = input_reshape[0] * input_reshape[2];
        unreduce_size = input_reshape[1];
      } else if ((input_reshape.size() == 3) && (!reduce_first_axis)) {
        reduce_size = input_reshape[1];
        unreduce_size = input_reshape[0] * input_reshape[2];
      }
      ReduceImpl<T, Sum<T>>(input, input_reshape, reduce_first_axis, Sum<T>(), output, hip_stream);
      Average<<<(unreduce_size + 256) / 256, 256, 0, hip_stream>>>(unreduce_size, reduce_size, output);
      break;
  }
  CHECK_CUDA_LAUNCH_SUCCESS();
}

template <typename T>
hipError_t ArrayReduceComplex(const T *input, const std::vector<size_t> &input_reshape, const bool reduce_first_axis,
                               ReduceType_t type, T *output, hipStream_t hip_stream) {
  switch (type) {
    case ReduceMax:
      break;
    case ReduceMin:
      break;
    case ReduceAll:
      break;
    case ReduceAny:
      break;
    case ReduceSum:
      ReduceImpl<T, Sum<T>>(input, input_reshape, reduce_first_axis, Sum<T>(), output, hip_stream);
      break;
    case ReduceProd:
      ReduceImpl<T, Prod<T>>(input, input_reshape, reduce_first_axis, Prod<T>(), output, hip_stream);
      break;
    case ReduceMean:
      size_t reduce_size = 1;
      size_t unreduce_size = 1;
      if (input_reshape.size() == 1) {
        reduce_size = input_reshape[0];
        unreduce_size = 1;
      } else if ((input_reshape.size() == 2) && (reduce_first_axis)) {
        reduce_size = input_reshape[0];
        unreduce_size = input_reshape[1];
      } else if ((input_reshape.size() == 2) && (!reduce_first_axis)) {
        reduce_size = input_reshape[1];
        unreduce_size = input_reshape[0];
      } else if ((input_reshape.size() == 3) && (reduce_first_axis)) {
        reduce_size = input_reshape[0] * input_reshape[2];
        unreduce_size = input_reshape[1];
      } else if ((input_reshape.size() == 3) && (!reduce_first_axis)) {
        reduce_size = input_reshape[1];
        unreduce_size = input_reshape[0] * input_reshape[2];
      }
      ReduceImpl<T, Sum<T>>(input, input_reshape, reduce_first_axis, Sum<T>(), output, hip_stream);
      Average<<<(unreduce_size + 256) / 256, 256, 0, hip_stream>>>(unreduce_size, reduce_size, output);
      break;
  }
  CHECK_CUDA_LAUNCH_SUCCESS();
}

#define ARRAY_REDUCE_REGISTER(T)                                                                               \
  template CUDA_LIB_EXPORT hipError_t ArrayReduce(const T *input, const std::vector<size_t> &input_reshape,   \
                                                   const bool reduce_first_axis, ReduceType_t type, T *output, \
                                                   hipStream_t hip_stream)

ARRAY_REDUCE_REGISTER(float);
ARRAY_REDUCE_REGISTER(double);
ARRAY_REDUCE_REGISTER(half);
ARRAY_REDUCE_REGISTER(bool);
ARRAY_REDUCE_REGISTER(int8_t);
ARRAY_REDUCE_REGISTER(int16_t);
ARRAY_REDUCE_REGISTER(int32_t);
ARRAY_REDUCE_REGISTER(int64_t);
ARRAY_REDUCE_REGISTER(uint8_t);
ARRAY_REDUCE_REGISTER(uint16_t);
ARRAY_REDUCE_REGISTER(uint32_t);
ARRAY_REDUCE_REGISTER(uint64_t);

#define ARRAY_REDUCE_COMPLEX_REGISTER(T)                                                                              \
  template CUDA_LIB_EXPORT hipError_t ArrayReduceComplex(const T *input, const std::vector<size_t> &input_reshape,   \
                                                          const bool reduce_first_axis, ReduceType_t type, T *output, \
                                                          hipStream_t hip_stream)

ARRAY_REDUCE_COMPLEX_REGISTER(Complex<float>);
ARRAY_REDUCE_COMPLEX_REGISTER(Complex<double>);
